
#include <hip/hip_runtime.h>
#include<bits/stdc++.h>

using namespace std;
const int MAX = 501;
bool visited[MAX][MAX][4];

int moveY[] = {-1, 0, 1, 0};
int moveX[] = {0, -1, 0, 1};

vector<int> solution(vector<string> grid) {
    vector<int> answer;

    int row = grid.size();
    int col = grid[0].size();

    auto BFS = [&](int y, int x, int dir) -> int {
        int count = 0;

        while(1){

            /**
             * 탐색 값(ex:)S, L, R)의 방향에 방문한적이 있다면 탐색을 종료한다.
             */
            if(visited[y][x][dir]){
                break;
            }
            visited[y][x][dir] = true;
            count++;

            /**
             * 방향 전환
             */

            if(grid[y][x] == 'L'){
                dir = dir ? dir - 1 : 3;
            }

            else if(grid[y][x] == 'R'){
                dir = dir == 3 ? 0 : dir + 1;
            }

            y = (y + moveY[dir] + row) % row;
            x = (x + moveX[dir] + col) % col;
        }
        return count;
    };

    /**
     * grid내의 모든 값들의 4가지 방향을 탐색해준다.
     */

    for (int i = 0; i < grid.size(); i++) {
        for (int j = 0; j < grid[i].size(); j++) {
            for(int k = 0; k < 4; k++) {
                if (!visited[i][j][k]) {
                    int count = BFS(i, j,k);
                    answer.push_back(count);
                }
            }
        }
    }
    sort(answer.begin(), answer.end());
    return answer;
}

int main() {
    vector<int> a = solution({"SL", "LR"});
//    vector<int> a = solution({"R,R"});

    for (const auto &item: a) {
        cout << item << " ";
    }
}